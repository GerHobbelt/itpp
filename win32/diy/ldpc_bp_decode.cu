
#include "ldpc_bp_decode.cuh"
#include "ldpc_bp_decode_kernel.cuh"

#include <hip/hip_runtime.h>
#include <thrust/reduce.h>
#include <thrust/device_vector.h>

bool syndrome_check_gpu(int *LLR, int nvar, 
	int* sumX2, int ncheck, 
	int* V, int nmaxX2 ) 
{
	// Please note the IT++ convention that a sure zero corresponds to
	// LLR=+infinity
	int* d_synd ;
	hipMalloc( (void**)&d_synd, ncheck * sizeof(int) );
	hipMemset( d_synd, 0, ncheck * sizeof(int) );

	int* d_LLR ;
	hipMalloc( (void**)&d_LLR, nvar * sizeof(int) );
	hipMemcpy( d_LLR, LLR, nvar * sizeof(int), hipMemcpyHostToDevice );

	int* d_sumX2 ;
	hipMalloc( (void**)&d_sumX2, ncheck * sizeof(int) );
	hipMemcpy( d_sumX2, sumX2, ncheck * sizeof(int), hipMemcpyHostToDevice );

	int* d_V ;
	hipMalloc( (void**)&d_V, ncheck * nmaxX2 * sizeof(int) );
	hipMemcpy( d_V, V, ncheck * nmaxX2 * sizeof(int), hipMemcpyHostToDevice );

	dim3 block( 256 );
	dim3 grid( (ncheck + block.x - 1) / block.x );

	syndrome_check_kernel<<< grid, block >>>( d_LLR, d_sumX2, ncheck, d_V, d_synd );

	int sum = thrust::reduce( thrust::device_ptr<int>( d_synd ),
		thrust::device_ptr<int>( d_synd + ncheck ), 
		(int) 0, thrust::plus<int>());

	hipFree( d_synd );
	hipFree( d_LLR );
	hipFree( d_sumX2 );
	hipFree( d_V );

	return sum == ncheck;   // codeword is valid
}

void updateVariableNode_gpu( int nvar, int ncheck, int nmaxX1, int nmaxX2, 
	int* sumX1, int* mcv, int* mvc, int* iind, int * LLRin, int * LLRout ) 
{

	int* d_sumX1 ;
	hipMalloc( (void**)&d_sumX1, nvar * sizeof(int) );
	hipMemcpy( d_sumX1, sumX1, nvar * sizeof(int), hipMemcpyHostToDevice );
	
	int* d_mcv ;
	hipMalloc( (void**)&d_mcv, ncheck * nmaxX2 * sizeof(int) );
	hipMemcpy( d_mcv, mcv, ncheck * nmaxX2 * sizeof(int), hipMemcpyHostToDevice );
		
	int* d_mvc ;
	hipMalloc( (void**)&d_mvc, nvar * nmaxX1 * sizeof(int) );
	hipMemcpy( d_mvc, mvc, nvar * nmaxX1 * sizeof(int), hipMemcpyHostToDevice );

	int* d_iind ;
	hipMalloc( (void**)&d_iind, nvar * nmaxX1 * sizeof(int) );
	hipMemcpy( d_iind, iind, nvar * nmaxX1 * sizeof(int), hipMemcpyHostToDevice );

	int* d_LLRin ;
	hipMalloc( (void**)&d_LLRin, nvar * sizeof(int) );
	hipMemcpy( d_LLRin, LLRin, nvar * sizeof(int), hipMemcpyHostToDevice );

	int* d_LLRout ;
	hipMalloc( (void**)&d_LLRout, nvar * sizeof(int) );
	hipMemcpy( d_LLRout, LLRout, nvar * sizeof(int), hipMemcpyHostToDevice );

	dim3 block( 256 );
	dim3 grid( (nvar + block.x - 1) / block.x );

	updateVariableNode_kernel<<< grid, block >>>( nvar, d_sumX1, d_mcv, d_mvc, d_iind, d_LLRin, d_LLRout );
	
	hipMemcpy( LLRout, d_LLRout, nvar * sizeof(int), hipMemcpyDeviceToHost );
	hipMemcpy( mvc, d_mvc, nvar * nmaxX1 * sizeof(int), hipMemcpyDeviceToHost );

	hipFree( d_sumX1 );
	hipFree( d_mcv );
	hipFree( d_mvc );
	hipFree( d_iind );
	hipFree( d_LLRin );
	hipFree( d_LLRout );

}

void updateCheckNode_gpu( int nvar, int ncheck, int nmaxX1, int nmaxX2, 
	int* sumX2, int* mcv, int* mvc, int* jind, 
	short int Dint1, short int Dint2, short int Dint3, int* logexp_table,
	int* jj, int* m, int* ml, int* mr, int max_cnd, int QLLR_MAX )
{
	
	int* d_sumX2 ;
	hipMalloc( (void**)&d_sumX2, ncheck * sizeof(int) );
	hipMemcpy( d_sumX2, sumX2, ncheck * sizeof(int), hipMemcpyHostToDevice );

	int* d_mcv ;
	hipMalloc( (void**)&d_mcv, ncheck * nmaxX2 * sizeof(int) );
	hipMemcpy( d_mcv, mcv, ncheck * nmaxX2 * sizeof(int), hipMemcpyHostToDevice );
		
	int* d_mvc ;
	hipMalloc( (void**)&d_mvc, nvar * nmaxX1 * sizeof(int) );
	hipMemcpy( d_mvc, mvc, nvar * nmaxX1 * sizeof(int), hipMemcpyHostToDevice );

	int* d_jind ;
	hipMalloc( (void**)&d_jind, nvar * nmaxX2 * sizeof(int) );
	hipMemcpy( d_jind, jind, ncheck * nmaxX2 * sizeof(int), hipMemcpyHostToDevice );
	
	int* d_logexp_table ;
	hipMalloc( (void**)&d_logexp_table, Dint2 * sizeof(int) );
	hipMemcpy( d_logexp_table, logexp_table, Dint2 * sizeof(int), hipMemcpyHostToDevice );
	
	int* d_jj ;
	hipMalloc( (void**)&d_jj, max_cnd * sizeof(int) );
	hipMemset( d_jj, 0, max_cnd * sizeof(int) );
	
	int* d_m ;
	hipMalloc( (void**)&d_m, max_cnd * sizeof(int) );
	hipMemset( d_m, 0, max_cnd * sizeof(int) );
	
	int* d_ml ;
	hipMalloc( (void**)&d_ml, max_cnd * sizeof(int) );
	hipMemset( d_ml, 0, max_cnd * sizeof(int) );
	
	int* d_mr ;
	hipMalloc( (void**)&d_mr, max_cnd * sizeof(int) );
	hipMemset( d_mr, 0, max_cnd * sizeof(int) );

	dim3 block( 256 );
	dim3 grid( (ncheck + block.x - 1) / block.x );

	updateCheckNode_kernel<<< grid, block >>>(ncheck, 
		d_sumX2, d_mcv, d_mvc, d_jind, Dint1, Dint2, Dint3, d_logexp_table,
		d_jj, d_m, d_ml, d_mr, QLLR_MAX );

	hipMemcpy( mcv, d_mcv, ncheck * nmaxX2 * sizeof(int), hipMemcpyDeviceToHost );
	hipMemcpy( mvc, d_mvc, nvar * nmaxX1 * sizeof(int), hipMemcpyDeviceToHost );

	hipFree( d_sumX2 );
	hipFree( d_mcv );
	hipFree( d_mvc );
	hipFree( d_jind );
	hipFree( d_logexp_table );
	hipFree( d_jj );	hipFree( d_m );	hipFree( d_ml );	hipFree( d_mr );
	
}