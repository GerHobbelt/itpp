
#include "ldpc_bp_decode.cuh"
#include "ldpc_bp_decode_kernel.cuh"

#include <hip/hip_runtime.h>
#include <thrust/reduce.h>
#include <thrust/device_vector.h>

bool syndrome_check_gpu(int *LLR, int nvar, 
	int* sumX2, int ncheck, 
	int* V, int nmaxX2 ) 
{
	// Please note the IT++ convention that a sure zero corresponds to
	// LLR=+infinity
	int* d_synd ;
	hipMalloc( (void**)&d_synd, ncheck * sizeof(int) );
	hipMemset( d_synd, 0, ncheck * sizeof(int) );

	int* d_LLR ;
	hipMalloc( (void**)&d_LLR, nvar * sizeof(int) );
	hipMemcpy( d_LLR, LLR, nvar * sizeof(int), hipMemcpyHostToDevice );

	int* d_sumX2 ;
	hipMalloc( (void**)&d_sumX2, ncheck * sizeof(int) );
	hipMemcpy( d_sumX2, sumX2, ncheck * sizeof(int), hipMemcpyHostToDevice );

	int* d_V ;
	hipMalloc( (void**)&d_V, ncheck * nmaxX2 * sizeof(int) );
	hipMemcpy( d_V, V, ncheck * nmaxX2 * sizeof(int), hipMemcpyHostToDevice );

	dim3 block( 256 );
	dim3 grid( (ncheck + block.x - 1) / block.x );

	syndrome_check_kernel<<< grid, block >>>( d_LLR, d_sumX2, ncheck, d_V, d_synd );

	int sum = thrust::reduce( thrust::device_ptr<int>( d_synd ),
		thrust::device_ptr<int>( d_synd + ncheck ), 
		(int) 0, thrust::plus<int>());

	hipFree( d_synd );
	hipFree( d_LLR );
	hipFree( d_sumX2 );
	hipFree( d_V );

	return sum == ncheck;   // codeword is valid
}

void updateVariableNode_gpu( int nvar, int ncheck, int nmaxX1, int nmaxX2, 
	int* sumX1, int* mcv, int* mvc, int* iind, int * LLRin, int * LLRout ) 
{

	int* d_sumX1 ;
	hipMalloc( (void**)&d_sumX1, nvar * sizeof(int) );
	hipMemcpy( d_sumX1, sumX1, nvar * sizeof(int), hipMemcpyHostToDevice );
	
	int* d_mcv ;
	hipMalloc( (void**)&d_mcv, ncheck * nmaxX2 * sizeof(int) );
	hipMemcpy( d_mcv, mcv, ncheck * nmaxX2 * sizeof(int), hipMemcpyHostToDevice );
		
	int* d_mvc ;
	hipMalloc( (void**)&d_mvc, nvar * nmaxX1 * sizeof(int) );
	hipMemcpy( d_mvc, mvc, nvar * nmaxX1 * sizeof(int), hipMemcpyHostToDevice );

	int* d_iind ;
	hipMalloc( (void**)&d_iind, nvar * nmaxX1 * sizeof(int) );
	hipMemcpy( d_iind, iind, nvar * nmaxX1 * sizeof(int), hipMemcpyHostToDevice );

	int* d_LLRin ;
	hipMalloc( (void**)&d_LLRin, nvar * sizeof(int) );
	hipMemcpy( d_LLRin, LLRin, nvar * sizeof(int), hipMemcpyHostToDevice );

	int* d_LLRout ;
	hipMalloc( (void**)&d_LLRout, nvar * sizeof(int) );
	hipMemcpy( d_LLRout, LLRout, nvar * sizeof(int), hipMemcpyHostToDevice );

	dim3 block( 256 );
	dim3 grid( (nvar + block.x - 1) / block.x );

	updateVariableNode_kernel<<< grid, block >>>( nvar, d_sumX1, d_mcv, d_mvc, d_iind, d_LLRin, d_LLRout );
	
	hipMemcpy( LLRout, d_LLRout, nvar * sizeof(int), hipMemcpyDeviceToHost );
	hipMemcpy( mvc, d_mvc, nvar * nmaxX1 * sizeof(int), hipMemcpyDeviceToHost );

	hipFree( d_sumX1 );
	hipFree( d_mcv );
	hipFree( d_mvc );
	hipFree( d_iind );
	hipFree( d_LLRin );
	hipFree( d_LLRout );

}

void updateCheckNode_gpu( int nvar, int ncheck, int nmaxX1, int nmaxX2, 
	int* sumX2, int* mcv, int* mvc, int* jind, 
	short int Dint1, short int Dint2, short int Dint3, int* logexp_table,
	int* jj, int* m, int* ml, int* mr, int QLLR_MAX )
{
	
	int* d_sumX2 ;
	hipMalloc( (void**)&d_sumX2, ncheck * sizeof(int) );
	hipMemcpy( d_sumX2, sumX2, ncheck * sizeof(int), hipMemcpyHostToDevice );

	int* d_mcv ;
	hipMalloc( (void**)&d_mcv, ncheck * nmaxX2 * sizeof(int) );
	hipMemcpy( d_mcv, mcv, ncheck * nmaxX2 * sizeof(int), hipMemcpyHostToDevice );
		
	int* d_mvc ;
	hipMalloc( (void**)&d_mvc, nvar * nmaxX1 * sizeof(int) );
	hipMemcpy( d_mvc, mvc, nvar * nmaxX1 * sizeof(int), hipMemcpyHostToDevice );

	int* d_jind ;
	hipMalloc( (void**)&d_jind, nvar * nmaxX2 * sizeof(int) );
	hipMemcpy( d_jind, jind, ncheck * nmaxX2 * sizeof(int), hipMemcpyHostToDevice );

	dim3 block( 256 );
	dim3 grid( (ncheck + block.x - 1) / block.x );

	updateCheckNode_kernel<<< grid, block >>>(ncheck, 
		sumX2, mcv, mvc, jind, Dint1, Dint2, Dint3, logexp_table,
		jj, m, ml, mr, QLLR_MAX );

}