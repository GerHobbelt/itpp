#include "hip/hip_runtime.h"

#include "ldpc_bp_decode.cuh"
#include "ldpc_bp_decode_kernel.cuh"

#include <hip/hip_runtime.h>
#include <thrust/reduce.h>
#include <thrust/device_vector.h>

bool ldpc_gpu::syndrome_check_gpu(int *LLR, int nvar, 
	int* sumX2, int ncheck, 
	int* V, int nmaxX2 ) 
{
	// Please note the IT++ convention that a sure zero corresponds to
	// LLR=+infinity
	int* d_synd ;
	hipMalloc( (void**)&d_synd, ncheck * sizeof(int) );
	hipMemset( d_synd, 0, ncheck * sizeof(int) );

	int* d_LLR ;
	hipMalloc( (void**)&d_LLR, nvar * sizeof(int) );
	hipMemcpy( d_LLR, LLR, nvar * sizeof(int), hipMemcpyHostToDevice );

	int* d_sumX2 ;
	hipMalloc( (void**)&d_sumX2, ncheck * sizeof(int) );
	hipMemcpy( d_sumX2, sumX2, ncheck * sizeof(int), hipMemcpyHostToDevice );

	int* d_V ;
	hipMalloc( (void**)&d_V, ncheck * nmaxX2 * sizeof(int) );
	hipMemcpy( d_V, V, ncheck * nmaxX2 * sizeof(int), hipMemcpyHostToDevice );

	dim3 block( 256 );
	dim3 grid( (ncheck + block.x - 1) / block.x );

	syndrome_check_kernel<<< grid, block >>>( d_LLR, d_sumX2, ncheck, d_V, d_synd );

	int sum = thrust::reduce( thrust::device_ptr<int>( d_synd ),
		thrust::device_ptr<int>( d_synd + ncheck ), 
		(int) 0, thrust::plus<int>());

	hipFree( d_synd );
	hipFree( d_LLR );
	hipFree( d_sumX2 );
	hipFree( d_V );

	return sum == ncheck;   // codeword is valid
}

void ldpc_gpu::updateVariableNode_gpu( int nvar, int ncheck, int nmaxX1, int nmaxX2, 
	int* sumX1, int* mcv, int* mvc, int* iind, int * LLRin, int * LLRout ) 
{

	int* d_sumX1 ;
	hipMalloc( (void**)&d_sumX1, nvar * sizeof(int) );
	hipMemcpy( d_sumX1, sumX1, nvar * sizeof(int), hipMemcpyHostToDevice );
	
	int* d_mcv ;
	hipMalloc( (void**)&d_mcv, ncheck * nmaxX2 * sizeof(int) );
	hipMemcpy( d_mcv, mcv, ncheck * nmaxX2 * sizeof(int), hipMemcpyHostToDevice );
		
	int* d_mvc ;
	hipMalloc( (void**)&d_mvc, nvar * nmaxX1 * sizeof(int) );
	hipMemcpy( d_mvc, mvc, nvar * nmaxX1 * sizeof(int), hipMemcpyHostToDevice );

	int* d_iind ;
	hipMalloc( (void**)&d_iind, nvar * nmaxX1 * sizeof(int) );
	hipMemcpy( d_iind, iind, nvar * nmaxX1 * sizeof(int), hipMemcpyHostToDevice );

	int* d_LLRin ;
	hipMalloc( (void**)&d_LLRin, nvar * sizeof(int) );
	hipMemcpy( d_LLRin, LLRin, nvar * sizeof(int), hipMemcpyHostToDevice );

	int* d_LLRout ;
	hipMalloc( (void**)&d_LLRout, nvar * sizeof(int) );
	hipMemcpy( d_LLRout, LLRout, nvar * sizeof(int), hipMemcpyHostToDevice );

	dim3 block( 256 );
	dim3 grid( (nvar + block.x - 1) / block.x );

	updateVariableNode_kernel<<< grid, block >>>( nvar, d_sumX1, d_mcv, d_mvc, d_iind, d_LLRin, d_LLRout );
	
	hipMemcpy( LLRout, d_LLRout, nvar * sizeof(int), hipMemcpyDeviceToHost );
	hipMemcpy( mvc, d_mvc, nvar * nmaxX1 * sizeof(int), hipMemcpyDeviceToHost );

	hipFree( d_sumX1 );
	hipFree( d_mcv );
	hipFree( d_mvc );
	hipFree( d_iind );
	hipFree( d_LLRin );
	hipFree( d_LLRout );

}

void ldpc_gpu::updateCheckNode_gpu( int nvar, int ncheck, int nmaxX1, int nmaxX2, 
	int* sumX2, int* mcv, int* mvc, int* jind, 
	short int Dint1, short int Dint2, short int Dint3, int* logexp_table,
	int* jj, int* m, int* ml, int* mr, int max_cnd, int QLLR_MAX )
{
	
	int* d_sumX2 ;
	hipMalloc( (void**)&d_sumX2, ncheck * sizeof(int) );
	hipMemcpy( d_sumX2, sumX2, ncheck * sizeof(int), hipMemcpyHostToDevice );

	int* d_mcv ;
	hipMalloc( (void**)&d_mcv, ncheck * nmaxX2 * sizeof(int) );
	hipMemcpy( d_mcv, mcv, ncheck * nmaxX2 * sizeof(int), hipMemcpyHostToDevice );
		
	int* d_mvc ;
	hipMalloc( (void**)&d_mvc, nvar * nmaxX1 * sizeof(int) );
	hipMemcpy( d_mvc, mvc, nvar * nmaxX1 * sizeof(int), hipMemcpyHostToDevice );

	int* d_jind ;
	hipMalloc( (void**)&d_jind, nvar * nmaxX2 * sizeof(int) );
	hipMemcpy( d_jind, jind, ncheck * nmaxX2 * sizeof(int), hipMemcpyHostToDevice );
	
	int* d_logexp_table ;
	hipMalloc( (void**)&d_logexp_table, Dint2 * sizeof(int) );
	hipMemcpy( d_logexp_table, logexp_table, Dint2 * sizeof(int), hipMemcpyHostToDevice );
	
	int* d_jj ;
	hipMalloc( (void**)&d_jj, ncheck * max_cnd * sizeof(int) );
	hipMemset( d_jj, 0, ncheck * max_cnd * sizeof(int) );
	
	int* d_m ;
	hipMalloc( (void**)&d_m, ncheck * max_cnd * sizeof(int) );
	hipMemset( d_m, 0, ncheck * max_cnd * sizeof(int) );
	
	int* d_ml ;
	hipMalloc( (void**)&d_ml, ncheck * max_cnd * sizeof(int) );
	hipMemset( d_ml, 0, ncheck * max_cnd * sizeof(int) );
	
	int* d_mr ;
	hipMalloc( (void**)&d_mr, ncheck * max_cnd * sizeof(int) );
	hipMemset( d_mr, 0, ncheck * max_cnd * sizeof(int) );

	dim3 block( 256 );
	dim3 grid( (ncheck + block.x - 1) / block.x );

	updateCheckNode_kernel<<< grid, block >>>(ncheck, 
		d_sumX2, d_mcv, d_mvc, d_jind, Dint1, Dint2, Dint3, d_logexp_table,
		d_jj, d_m, d_ml, d_mr, max_cnd, QLLR_MAX );

	hipMemcpy( mcv, d_mcv, ncheck * nmaxX2 * sizeof(int), hipMemcpyDeviceToHost );
	hipMemcpy( mvc, d_mvc, nvar * nmaxX1 * sizeof(int), hipMemcpyDeviceToHost );

	hipFree( d_sumX2 );
	hipFree( d_mcv );
	hipFree( d_mvc );
	hipFree( d_jind );
	hipFree( d_logexp_table );
	hipFree( d_jj );	hipFree( d_m );	hipFree( d_ml );	hipFree( d_mr );
	
}


int ldpc_gpu::bp_decode(int *LLRin, int *LLRout,
	int nvar, int ncheck, 
	int nmaxX1, int nmaxX2, // max(sumX1) max(sumX2)
	int* V, int* sumX1, int* sumX2, int* iind, int* jind,	// Parity check matrix parameterization
	int* mvc, int* mcv,	// temporary storage for decoder (memory allocated when codec defined)
	//LLR_calc_unit& llrcalc,		//!< LLR calculation unit
	short int Dint1, short int Dint2, short int Dint3,	//! Decoder (lookup-table) parameters
	int* logexp_table,		//! The lookup tables for the decoder
	bool psc /*= true*/,			//!< check syndrom after each iteration
	int max_iters /*= 50*/ )		//!< Maximum number of iterations
{

  // initial step
  for (int i = 0; i < nvar; i++) {
    int index = i;
    for (int j = 0; j < sumX1[i]; j++) {
      mvc[index] = LLRin[i];
      index += nvar;
    }
  }

  const int QLLR_MAX = (std::numeric_limits<int>::max() >> 4);

  //! Maximum check node degree that the class can handle
  static const int max_cnd = 200;

  // allocate temporary variables used for the check node update
  int jj[max_cnd];
  int m[max_cnd];
  int ml[max_cnd];
  int mr[max_cnd];


  bool is_valid_codeword = false;
  int iter = 0;
  do {
    iter++;
    //if (nvar >= 100000) { it_info_no_endl_debug("."); }
    // --------- Step 1: check to variable nodes ----------
	updateCheckNode_gpu(nvar, ncheck, nmaxX1, nmaxX2, 
		sumX2, mcv, mvc, jind, Dint1, Dint2, Dint3, logexp_table,
		jj, m, ml, mr, max_cnd, QLLR_MAX );

    // step 2: variable to check nodes
	updateVariableNode_gpu(nvar, ncheck, nmaxX1, nmaxX2, 
		sumX1, mcv, mvc, iind, LLRin, LLRout);

	if (psc && syndrome_check_gpu(LLRout, nvar, sumX2, ncheck, V, nmaxX2)) {
	  is_valid_codeword = true;
      break;
    }
  }
  while (iter < max_iters);

  return (is_valid_codeword ? iter : -iter);
}