
#include "ldpc_bp_decode.cuh"
#include "ldpc_bp_decode_kernel.cuh"

#include <hip/hip_runtime.h>
#include <thrust/reduce.h>
#include <thrust/device_vector.h>

bool syndrome_check_gpu(int *LLR, int nvar, 
	int* sumX2, int ncheck, 
	int* V, int maxX2 ) 
{
	// Please note the IT++ convention that a sure zero corresponds to
	// LLR=+infinity
	int* d_synd ;
	hipMalloc( (void**)&d_synd, ncheck * sizeof(int) );
	hipMemset( d_synd, 0, ncheck * sizeof(int) );

	int* d_LLR ;
	hipMalloc( (void**)&d_LLR, nvar * sizeof(int) );
	hipMemcpy( d_LLR, LLR, nvar * sizeof(int), hipMemcpyHostToDevice );

	int* d_sumX2 ;
	hipMalloc( (void**)&d_sumX2, ncheck * sizeof(int) );
	hipMemcpy( d_sumX2, sumX2, ncheck * sizeof(int), hipMemcpyHostToDevice );

	int* d_V ;
	hipMalloc( (void**)&d_V, ncheck * maxX2 * sizeof(int) );
	hipMemcpy( d_V, V, ncheck * maxX2 * sizeof(int), hipMemcpyHostToDevice );

	dim3 block( 256 );
	dim3 grid( (ncheck + block.x - 1) / block.x );

	syndrome_check_kernel<<< grid, block >>>( d_LLR, d_sumX2, ncheck, d_V, d_synd );

	int bValid = thrust::reduce( thrust::device_ptr<int>( d_synd ),
		thrust::device_ptr<int>( d_synd + ncheck ), 
		(int) 0, thrust::multiplies<int>());

	hipFree( d_synd );
	hipFree( d_LLR );
	hipFree( d_sumX2 );
	hipFree( d_V );

	return bValid;   // codeword is valid
}
