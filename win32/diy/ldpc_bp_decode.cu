#include "hip/hip_runtime.h"

#include "ldpc_bp_decode.cuh"
#include "ldpc_bp_decode_kernel.cuh"

#include <hip/hip_runtime.h>
#include <thrust/reduce.h>
#include <thrust/device_vector.h>

bool ldpc_gpu::syndrome_check_gpu() 
{
	dim3 block( 256 );
	dim3 grid( (ncheck + block.x - 1) / block.x );

	syndrome_check_kernel<<< grid, block >>>( d_LLRout, d_sumX2, ncheck, d_V, d_synd );

	int sum = thrust::reduce( thrust::device_ptr<int>( d_synd ),
		thrust::device_ptr<int>( d_synd + ncheck ), 
		(int) 0, thrust::plus<int>());

	return sum == ncheck;   // codeword is valid
}

void ldpc_gpu::updateVariableNode_gpu() 
{
	dim3 block( 256 );
	dim3 grid( (nvar + block.x - 1) / block.x );

	updateVariableNode_kernel<<< grid, block >>>( nvar, d_sumX1, d_mcv, d_mvc, d_iind, d_LLRin, d_LLRout );
}

void ldpc_gpu::updateCheckNode_gpu()
{
	dim3 block( 256 );
	dim3 grid( (ncheck + block.x - 1) / block.x );

	updateCheckNode_kernel<<< grid, block >>>(ncheck, 
		d_sumX2, d_mcv, d_mvc, d_jind, Dint1, Dint2, Dint3,
		d_jj, d_m, d_ml, d_mr, max_cnd, QLLR_MAX );
}

void ldpc_gpu::initializeMVC_gpu( )
{
	dim3 block( 256 );
	dim3 grid( (nvar + block.x - 1) / block.x );

	initializeMVC_kernel<<< grid, block >>>( nvar, d_sumX1, d_mvc, d_LLRin );
}

int ldpc_gpu::bp_decode(int *LLRin, int *LLRout,
	int* sumX1,	int* mvc, 
	bool psc /*= true*/,			//!< check syndrom after each iteration
	int max_iters /*= 50*/ )		//!< Maximum number of iterations
{
	hipMemcpy( d_LLRin, LLRin, nvar * sizeof(int), hipMemcpyHostToDevice );

  // initial step
	initializeMVC_gpu();

  bool is_valid_codeword = false;
  int iter = 0;
  do {
    iter++;
    //if (nvar >= 100000) { it_info_no_endl_debug("."); }
    // --------- Step 1: check to variable nodes ----------
	updateCheckNode_gpu();

    // step 2: variable to check nodes
	updateVariableNode_gpu();

	if (psc && syndrome_check_gpu()) {
	  is_valid_codeword = true;
      break;
    }
  }
  while (iter < max_iters);

  hipMemcpy( LLRout, d_LLRout, nvar * sizeof(int), hipMemcpyDeviceToHost );


  return (is_valid_codeword ? iter : -iter);
}

bool ldpc_gpu::initialize( int nvar, int ncheck,
	int nmaxX1, int nmaxX2,
	int* sumX1, int* sumX2, int* iind, int* jind, int* V, 	// Parity check matrix parameterization
	int* mvc, int* mcv,	// temporary storage for decoder (memory allocated when codec defined)
	short int Dint1, short int Dint2, short int Dint3,
	int* logexp_table		//! The lookup tables for the decoder
	)
{
	this->nvar = nvar;		this->ncheck = ncheck;
	this->nmaxX1 = nmaxX1;	this->nmaxX2 = nmaxX2; // max(sumX1) max(sumX2)
	this->Dint1 = Dint1;	this->Dint2 = Dint2;	this->Dint3 = Dint3;	//! Decoder (lookup-table) parameters
	
	max_cnd = 200;
	QLLR_MAX = (std::numeric_limits<int>::max() >> 4);

	hipMalloc( (void**)&d_LLRin, nvar * sizeof(int) );
	hipMalloc( (void**)&d_LLRout, nvar * sizeof(int) );
	hipMemset( d_LLRout, 0, nvar * sizeof(int) );

	hipMalloc( (void**)&d_synd, ncheck * sizeof(int) );
	hipMemset( d_synd, 0, ncheck * sizeof(int) );
	
	hipMalloc( (void**)&d_sumX1, nvar * sizeof(int) );		// const 64 K
	hipMemcpy( d_sumX1, sumX1, nvar * sizeof(int), hipMemcpyHostToDevice );

	hipMalloc( (void**)&d_sumX2, ncheck * sizeof(int) );	// const 32 K
	hipMemcpy( d_sumX2, sumX2, ncheck * sizeof(int), hipMemcpyHostToDevice );

	hipMalloc( (void**)&d_iind, nvar * nmaxX1 * sizeof(int) );		// const 1.2 M
	hipMemcpy( d_iind, iind, nvar * nmaxX1 * sizeof(int), hipMemcpyHostToDevice );
	
	hipMalloc( (void**)&d_jind, ncheck * nmaxX2 * sizeof(int) );	// const 300 K
	hipMemcpy( d_jind, jind, ncheck * nmaxX2 * sizeof(int), hipMemcpyHostToDevice );

	hipMalloc( (void**)&d_V, ncheck * nmaxX2 * sizeof(int) );		// const 300 K
	hipMemcpy( d_V, V, ncheck * nmaxX2 * sizeof(int), hipMemcpyHostToDevice );
	
	hipMalloc( (void**)&d_mcv, ncheck * nmaxX2 * sizeof(int) );
	hipMemcpy( d_mcv, mcv, ncheck * nmaxX2 * sizeof(int), hipMemcpyHostToDevice );
		
	hipMalloc( (void**)&d_mvc, nvar * nmaxX1 * sizeof(int) );
	hipMemcpy( d_mvc, mvc, nvar * nmaxX1 * sizeof(int), hipMemcpyHostToDevice );

	//hipMalloc( (void**)&d_logexp_table, Dint2 * sizeof(int) );		// const 1.2 K
	//hipMemcpy( d_logexp_table, logexp_table, Dint2 * sizeof(int), hipMemcpyHostToDevice );

	initConstantMemory(logexp_table);

	hipMalloc( (void**)&d_jj, ncheck * max_cnd * sizeof(int) );
	hipMemset( d_jj, 0, ncheck * max_cnd * sizeof(int) );
	
	hipMalloc( (void**)&d_m, ncheck * max_cnd * sizeof(int) );
	hipMemset( d_m, 0, ncheck * max_cnd * sizeof(int) );
	
	hipMalloc( (void**)&d_ml, ncheck * max_cnd * sizeof(int) );
	hipMemset( d_ml, 0, ncheck * max_cnd * sizeof(int) );
	
	hipMalloc( (void**)&d_mr, ncheck * max_cnd * sizeof(int) );
	hipMemset( d_mr, 0, ncheck * max_cnd * sizeof(int) );

	return true;
}


bool ldpc_gpu::release()
{
	hipFree( d_LLRin );	hipFree( d_LLRout );
	
	hipFree( d_synd );

	hipFree( d_sumX1 );	hipFree( d_sumX2 );
	
	hipFree( d_iind );		hipFree( d_jind );
	hipFree( d_V );

	hipFree( d_mcv );		hipFree( d_mvc );
	
	//hipFree( d_logexp_table );	

	hipFree( d_jj );	hipFree( d_m );	hipFree( d_ml );	hipFree( d_mr );

	return true;
}

ldpc_gpu::~ldpc_gpu()
{
	release();
}