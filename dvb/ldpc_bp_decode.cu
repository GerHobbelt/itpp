#include "hip/hip_runtime.h"

#include "ldpc_bp_decode.cuh"
#include "ldpc_bp_decode_kernel.cuh"

#include <hip/hip_runtime.h>
#include <thrust/reduce.h>
#include <thrust/device_vector.h>

#if USE_TEXTURE_ADDRESS
	hipArray* arr_mcv;
	hipArray* arr_mvc;
	hipChannelFormatDesc channelDesc;
#endif

bool ldpc_gpu::syndrome_check_gpu() 
{
	dim3 block( SIZE_BLOCK );
	dim3 grid( (ncheck + block.x - 1) / block.x );

	syndrome_check_kernel<<< grid, block >>>( d_LLRout, d_sumX2, ncheck, d_V, d_synd );

	int h_synd=0;
	hipMemcpy( &h_synd, d_synd, sizeof(int), hipMemcpyDeviceToHost );

	return h_synd == 0;   // codeword is valid
}

void ldpc_gpu::updateVariableNode_gpu() 
{
	dim3 block( SIZE_BLOCK );
	dim3 grid( (nvar + block.x - 1) / block.x );

	updateVariableNode_kernel<<< grid, block >>>( nvar, ncheck, d_sumX1, d_mcv, d_iind, d_LLRin, d_LLRout, d_mvc, d_bLLR );
}

void ldpc_gpu::updateCheckNode_gpu()
{
	dim3 block( SIZE_BLOCK );
	dim3 grid( (ncheck + block.x - 1) / block.x );

	updateCheckNode_kernel<<< grid, block >>>(ncheck, nvar, 
		d_sumX2, d_mvc, d_jind, Dint1, Dint2, Dint3,
		d_ml, d_mr, max_cnd, QLLR_MAX, d_mcv );	// Shared not faster
}

void ldpc_gpu::initializeMVC_gpu( )
{
	dim3 block( 256 );
	dim3 grid( (nvar + block.x - 1) / block.x );

	initializeMVC_kernel<<< grid, block >>>( nvar, d_sumX1, d_LLRin, d_mvc );
}

int ldpc_gpu::bp_decode(int *LLRin, int *LLRout,
	bool psc /*= true*/,			//!< check syndrom after each iteration
	int max_iters /*= 50*/ )		//!< Maximum number of iterations
{
	hipMemcpy( d_LLRin, LLRin, nvar * sizeof(int), hipMemcpyHostToDevice );

  // initial step
	initializeMVC_gpu();

  bool is_valid_codeword = false;
  int iter = 0;
  do {
    iter++;
    //if (nvar >= 100000) { it_info_no_endl_debug("."); }
    // --------- Step 1: check to variable nodes ----------
	updateCheckNode_gpu();

#if USE_TEXTURE_ADDRESS
    // update the array to the texture
    hipMemcpyToArray(arr_mcv, 0, 0, d_mcv, ncheck * nmaxX2 * sizeof(int), hipMemcpyDeviceToDevice);
#endif

    // step 2: variable to check nodes
	updateVariableNode_gpu();

#if USE_TEXTURE_ADDRESS
    // update the array to the texture
    hipMemcpyToArray(arr_mvc, 0, 0, d_mvc, nvar * nmaxX1 * sizeof(int), hipMemcpyDeviceToDevice);
#endif

#if	USE_TABLE_CODE
	updateConstantMemoryLLRByte( d_bLLR );
#endif

	if (psc && syndrome_check_gpu()) {
	  is_valid_codeword = true;
      break;
    }
  }
  while (iter < max_iters);

  hipMemcpy( LLRout, d_LLRout, nvar * sizeof(int), hipMemcpyDeviceToHost );


  return (is_valid_codeword ? iter : -iter);
}

int ldpc_gpu::bp_decode_once(int *LLRin, int *LLRout,
	bool psc /*= true*/,			//!< check syndrom after each iteration
	int max_iters /*= 50*/ )		//!< Maximum number of iterations
{
	hipMemcpy( d_LLRin, LLRin, nvar * sizeof(int), hipMemcpyHostToDevice );

  // initial step
	initializeMVC_gpu();

  bool is_valid_codeword = false;
  int iter = 0;
  do {
    iter++;

	updateCheckNode_gpu();

    // step 2: variable to check nodes
	updateVariableNode_gpu();

	is_valid_codeword = syndrome_check_gpu();

	if ( is_valid_codeword ) {
      break;
    }
  }
  while (iter < max_iters);

  hipMemcpy( LLRout, d_LLRout, nvar * sizeof(int), hipMemcpyDeviceToHost );


  return (is_valid_codeword ? iter : -iter);
}

bool ldpc_gpu::initialize( int nvar, int ncheck,
	int nmaxX1, int nmaxX2,
	int* sumX1, int* sumX2, int* iind, int* jind, int* V, 	// Parity check matrix parameterization
	int* mvc, int* mcv,	// temporary storage for decoder (memory allocated when codec defined)
	short int Dint1, short int Dint2, short int Dint3,
	int* logexp_table		//! The lookup tables for the decoder
	)
{
	this->nvar = nvar;		this->ncheck = ncheck;
	this->nmaxX1 = nmaxX1;	this->nmaxX2 = nmaxX2; // max(sumX1) max(sumX2)
	this->Dint1 = Dint1;	this->Dint2 = Dint2;	this->Dint3 = Dint3;	//! Decoder (lookup-table) parameters
	
	max_cnd = 200;
	QLLR_MAX = (std::numeric_limits<int>::max() >> 4);

	hipMalloc( (void**)&d_LLRin, nvar * sizeof(int) );
	hipMalloc( (void**)&d_LLRout, nvar * sizeof(int) );
	hipMemset( d_LLRout, 0, nvar * sizeof(int) );

	hipMalloc( (void**)&d_bLLR, nvar * sizeof(char) );

	hipMalloc( (void**)&d_synd, 1 * sizeof(int) );
	hipMemset( d_synd, 0, 1 * sizeof(int) );
	
	hipMalloc( (void**)&d_sumX1, nvar * sizeof(int) );		// const 64 K
	hipMemcpy( d_sumX1, sumX1, nvar * sizeof(int), hipMemcpyHostToDevice );

	hipMalloc( (void**)&d_sumX2, ncheck * sizeof(int) );	// const 32 K
	hipMemcpy( d_sumX2, sumX2, ncheck * sizeof(int), hipMemcpyHostToDevice );

	hipMalloc( (void**)&d_iind, nvar * nmaxX1 * sizeof(int) );		// const 1.2 M
	hipMemcpy( d_iind, iind, nvar * nmaxX1 * sizeof(int), hipMemcpyHostToDevice );
	
	hipMalloc( (void**)&d_jind, ncheck * nmaxX2 * sizeof(int) );	// const 300 K
	hipMemcpy( d_jind, jind, ncheck * nmaxX2 * sizeof(int), hipMemcpyHostToDevice );

	hipMalloc( (void**)&d_V, ncheck * nmaxX2 * sizeof(int) );		// const 300 K
	hipMemcpy( d_V, V, ncheck * nmaxX2 * sizeof(int), hipMemcpyHostToDevice );
	
	hipMalloc( (void**)&d_mcv, ncheck * nmaxX2 * sizeof(int) );
	hipMemcpy( d_mcv, mcv, ncheck * nmaxX2 * sizeof(int), hipMemcpyHostToDevice );
		
	hipMalloc( (void**)&d_mvc, nvar * nmaxX1 * sizeof(int) );
	hipMemcpy( d_mvc, mvc, nvar * nmaxX1 * sizeof(int), hipMemcpyHostToDevice );

	//hipMalloc( (void**)&d_logexp_table, Dint2 * sizeof(int) );		// const 1.2 K
	//hipMemcpy( d_logexp_table, logexp_table, Dint2 * sizeof(int), hipMemcpyHostToDevice );

	initConstantMemoryLogExp(logexp_table);
	
	hipMalloc( (void**)&d_ml, ncheck * max_cnd * sizeof(int) );
	hipMemset( d_ml, 0, ncheck * max_cnd * sizeof(int) );
	
	hipMalloc( (void**)&d_mr, ncheck * max_cnd * sizeof(int) );
	hipMemset( d_mr, 0, ncheck * max_cnd * sizeof(int) );

#if USE_TEXTURE_ADDRESS
	// cuda texture ------------------------------------------------------------------------------------------
	channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindSigned);
    hipError_t err = hipMallocArray(&arr_mcv, &channelDesc, ncheck, nmaxX2);
    hipMemcpyToArray(arr_mcv, 0, 0, d_mcv, ncheck * nmaxX2 * sizeof(int), hipMemcpyDeviceToDevice);

	texMCV.addressMode[0] = hipAddressModeClamp;
	texMCV.addressMode[1] = hipAddressModeClamp;
    texMCV.filterMode = hipFilterModePoint;
    texMCV.normalized = false;

	hipBindTextureToArray(texMCV, arr_mcv, channelDesc);

	hipMallocArray(&arr_mvc, &channelDesc, nvar, nmaxX1);
    hipMemcpyToArray(arr_mvc, 0, 0, d_mvc, nvar * nmaxX1 * sizeof(int), hipMemcpyDeviceToDevice);
	hipBindTextureToArray(texMVC, arr_mvc, channelDesc);

#endif

	return true;
}


bool ldpc_gpu::release()
{
	hipFree( d_LLRin );	hipFree( d_LLRout );	hipFree( d_bLLR );
	
	hipFree( d_synd );

	hipFree( d_sumX1 );	hipFree( d_sumX2 );
	
	hipFree( d_iind );		hipFree( d_jind );
	hipFree( d_V );

	hipFree( d_mcv );		hipFree( d_mvc );
	
	//hipFree( d_logexp_table );	

	hipFree( d_ml );	hipFree( d_mr );

	return true;
}

ldpc_gpu::~ldpc_gpu()
{
	release();
}